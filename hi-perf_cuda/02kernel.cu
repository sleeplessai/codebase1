#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <memory>

#include "hip/hip_runtime_api.h"

#define Accesss_gpu_addr

__global__ void kernel(int* pret) {
  *pret = 0x2028;
}

int main(int argc, char const* argv[]) {
#ifndef Accesss_gpu_addr
  int ret = 0;
  int* hret = static_cast<int*>(malloc(sizeof(int)));

  kernel<<<1, 1>>>(&ret);
  kernel<<<1, 1>>>(hret);
  hipError_t err = hipDeviceSynchronize();
  printf("%d\n", ret);
  printf("%s : %s\n", hipGetErrorName(err), hipGetErrorString(err));
  // an illegal memory access was encountered

  checkCudaErrors(hipDeviceSynchronize());
  free(hret);
#else
  int* pret;
  checkCudaErrors(hipMalloc(&pret, sizeof(int)));
  kernel<<<1, 1>>>(pret);
  checkCudaErrors(hipDeviceSynchronize());
  // printf("ret_on_gpu: %d\n", *pret);

  int pdst;
  checkCudaErrors(hipMemcpy(&pdst, pret, sizeof(int), hipMemcpyDeviceToHost));
  // has once hipDeviceSynchronize()

  // hipMemcpyKind
  printf("dst_on_cpu: %d\n", pdst);

  hipFree(pret);
#endif

  return 0;
}
